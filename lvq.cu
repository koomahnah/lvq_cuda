
#include <hip/hip_runtime.h>
#include<cstdio>
#include <cassert>

#define max(a,b) ((a)>(b)?(a):(b))
#define min(a,b) ((a)<(b)?(a):(b))

#define THREADS_PER_BLOCK 32
extern "C" {
#define weight_index(neuron, i) ((neuron) * (input_dim) + (i))
__global__
void distance(int input_dim, float *neuron_weight, int *text_array, int text_index,
              float *neuron_dist)
{
    __shared__ float tab[THREADS_PER_BLOCK];
    int offset;
    int lx = threadIdx.x;
    float *this_neuron = neuron_weight + (blockIdx.x * input_dim);
    int *this_text = text_array + (text_index * input_dim);

    tab[lx] = 0;
    for (int i = 0; i < 8; i++) {
        int id = lx + i * THREADS_PER_BLOCK;
        if (id < input_dim) {
            if (i > 0)
                printf("reducing\n");
            printf("neuron%d: at index%d, this_neuron[%d]=%f, this_text[%d]=%d\n",
                    blockIdx.x, id, id, this_neuron[id], id, this_text[id]);
            float val = this_neuron[id] - this_text[id];
            tab[lx] += val * val;
        }
    }
    __syncthreads();
    for (offset = 1; offset < THREADS_PER_BLOCK; offset <<= 1) {
        float tmp = tab[lx];
        if (lx+offset < THREADS_PER_BLOCK)
            tmp = tmp + tab[lx+offset];
        __syncthreads();
        tab[lx] = tmp;
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        neuron_dist[blockIdx.x] = tab[0];
        printf("distance of neuron%d is %f\n", blockIdx.x, tab[0]);
    }
}
__global__
void init(int input_dim, int output_dim, int neuron_count, int *neuron_class,
          float *neuron_weight, int *neuron_bias)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    float val = (thid == 0) ? 0.3 : 0.5;
    
    neuron_class[thid] = thid * output_dim / neuron_count;
    for (i = 0; i < input_dim; i++)
        neuron_weight[weight_index(thid, i)] = val;
    neuron_bias[thid] = 0;

}
}

