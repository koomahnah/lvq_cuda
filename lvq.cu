
#include <hip/hip_runtime.h>
#include<cstdio>
#include <cassert>

#define max(a,b) ((a)>(b)?(a):(b))
#define min(a,b) ((a)<(b)?(a):(b))

#define THREADS_PER_BLOCK 32
#define ATTRACT_STEP 0.9

extern "C" {
#define weight_index(neuron, i) ((neuron) * (input_dim) + (i))
__global__
void attract(int input_dim, double *neuron_weight, int neuron_index,
        int *text_array, int text_index, double step)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid >= input_dim)
        return;

    double delta = step *
        (text_array[text_index * input_dim + thid] - neuron_weight[neuron_index * input_dim + thid]);
    neuron_weight[neuron_index * input_dim + thid] += delta;
}

__global__
void distance(int input_dim, double *neuron_weight, int *text_array, int text_index,
              double *neuron_dist)
{
    __shared__ double tab[THREADS_PER_BLOCK];
    int offset;
    int lx = threadIdx.x;
    double *this_neuron = neuron_weight + (blockIdx.x * input_dim);
    int *this_text = text_array + (text_index * input_dim);

    tab[lx] = 0;
    for (int i = 0; i < 8; i++) {
        int id = lx + i * THREADS_PER_BLOCK;
        if (id < input_dim) {
//            if (i > 0)
//                printf("reducing\n");
//            printf("neuron%d: at index%d, this_neuron[%d]=%f, this_text[%d]=%d\n",
//                    blockIdx.x, id, id, this_neuron[id], id, this_text[id]);
            double val = this_neuron[id] - this_text[id];
            tab[lx] += val * val;
        }
    }
    __syncthreads();
    for (offset = 1; offset < THREADS_PER_BLOCK; offset <<= 1) {
        double tmp = tab[lx];
        if (lx+offset < THREADS_PER_BLOCK)
            tmp = tmp + tab[lx+offset];
        __syncthreads();
        tab[lx] = tmp;
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        neuron_dist[blockIdx.x] = tab[0];
//        printf("distance of neuron%d is %f\n", blockIdx.x, tab[0]);
    }
}
__global__
void init(int input_dim, int output_dim, int neuron_count,
          double *neuron_weight)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    double val = (thid == 0) ? 0.3 : 0.5;
    
    for (i = 0; i < input_dim; i++)
        neuron_weight[weight_index(thid, i)] = val;

}
}

